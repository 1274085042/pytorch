#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

constexpr int64_t BYTES_PER_THREAD = 16;
constexpr int64_t MAX_NUM_THREADS = 1024;
constexpr int64_t MIN_NUM_THREADS = 128;
constexpr int64_t WARP_SIZE = 32;
constexpr int64_t BLOCK_SIZE = 32;

template <typename T>
__device__ inline void streamLoad128(uint4& val, const T* addr) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  asm("ld.global.nc.v2.u64 {%0, %1}, [%2];"
      : "=l"(low), "=l"(high)
      : "l"(addr));
  reinterpret_cast<unsigned long long int*>(&val)[0] = low;
  reinterpret_cast<unsigned long long int*>(&val)[1] = high;
#endif
}

template <typename T>
__device__ inline void streamStore128(T* addr, const uint4& val) {
#if defined(USE_ROCM) || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 800))
  CUDA_KERNEL_ASSERT(false);
#else
  unsigned long long int low, high;
  low = reinterpret_cast<const unsigned long long int*>(&val)[0];
  high = reinterpret_cast<const unsigned long long int*>(&val)[1];
  asm("st.global.cs.v2.u64 [%0], {%1, %2};" : : "l"(addr), "l"(low), "l"(high));
#endif
}

static __host__ __device__ inline int64_t divUp(int64_t a, int64_t b) {
  return (a + b - 1) / b;
}

static __host__ __device__ inline int64_t minInt64(int64_t a, int64_t b) {
  return a < b ? a : b;
}

static __host__ __device__ inline int64_t maxInt64(int64_t a, int64_t b) {
  return a < b ? b : a;
}

static __device__ inline bool isAligned(const void* ptr, size_t alignment) {
  uintptr_t addr = reinterpret_cast<uintptr_t>(ptr);
  return addr % alignment == 0;
}

__device__ inline uint4 initialize() {
  uint4 zero;
  reinterpret_cast<uint64_t*>(&zero)[0] = 0;
  reinterpret_cast<uint64_t*>(&zero)[1] = 0;
  return zero;
}

static __global__ void fsdpAllGatherCopyOutKernel(
    void** paramPtrs,
    void* allGatherResPtr,
    int64_t* blockOffsetToParamIdx,
    int64_t* blockCumSums,
    int64_t* shardDimCumSums,
    int64_t numBytesPerRank,
    int64_t numBlocksPerRank,
    int64_t rankStride,
    int64_t worldSize) {
  const int64_t blockOffset = blockIdx.x % numBlocksPerRank;
  const int64_t paramIdx = blockOffsetToParamIdx[blockOffset];
  for (int64_t rank = blockIdx.x / numBlocksPerRank; rank < worldSize;
       rank += rankStride) {
    const int64_t shardBlockCount =
        blockCumSums[paramIdx + 1] - blockCumSums[paramIdx];
    const int64_t groupSize = shardBlockCount * blockDim.x;
    const int64_t localTid =
        (blockOffset - blockCumSums[paramIdx]) * blockDim.x + threadIdx.x;

    const int64_t shardBegin = shardDimCumSums[paramIdx];
    const int64_t shardEnd = shardDimCumSums[paramIdx + 1];
    const int64_t shardLen = shardEnd - shardBegin;
    const int64_t srcOff = rank * numBytesPerRank + shardBegin;
    const int64_t dstOff = rank * shardLen;

    const char* srcPtr = reinterpret_cast<char*>(allGatherResPtr) + srcOff;
    char* dstPtr = reinterpret_cast<char*>(paramPtrs[paramIdx]) + dstOff;

    const int64_t alignOff =
        divUp(dstOff, BYTES_PER_THREAD) * BYTES_PER_THREAD - dstOff;
    const int64_t begin = alignOff + localTid * BYTES_PER_THREAD;
    const int64_t end =
        alignOff + (shardLen - alignOff) / BYTES_PER_THREAD * BYTES_PER_THREAD;
    const int64_t stride = groupSize * BYTES_PER_THREAD;

    for (size_t i = begin; i < end; i += stride) {
      uint4 val;
      if (isAligned(srcPtr + i, BYTES_PER_THREAD)) {
        streamLoad128(val, srcPtr + i);
      } else {
        for (size_t j = 0; j < BYTES_PER_THREAD; ++j) {
          reinterpret_cast<char*>(&val)[j] = srcPtr[i + j];
        }
      }
      streamStore128(&dstPtr[i], val);
    }
    if (localTid < alignOff && localTid < shardLen) {
      dstPtr[localTid] = srcPtr[localTid];
    }
    if (end + localTid < shardLen) {
      dstPtr[end + localTid] = srcPtr[end + localTid];
    }
  }
}

static int64_t geometricMean(const std::vector<int64_t>& numbers) {
  TORCH_CHECK(numbers.size() > 0);
  double logSum = 0.0;
  for (double num : numbers) {
    TORCH_CHECK(num > 0);
    logSum += log(num);
  }
  double avgLog = logSum / numbers.size();
  return exp(avgLog);
}

std::pair<at::Tensor, std::vector<int64_t*>> pack(
    std::vector<std::vector<int64_t>> vecs,
    const at::Device& device) {
  int64_t numel = 0;
  for (const auto& vec : vecs) {
    numel += vec.size();
  }

  auto packed = at::empty(
      {numel}, at::TensorOptions().dtype(at::kLong).pinned_memory(true));
  size_t offset = 0;
  for (const auto& vec : vecs) {
    memcpy(
        packed.data_ptr<int64_t>() + offset,
        vec.data(),
        sizeof(int64_t) * vec.size());
    offset += vec.size();
  }
  packed = packed.to(device, /*non_blocking=*/true);

  std::vector<int64_t*> ptrs;
  offset = 0;
  for (const auto& vec : vecs) {
    ptrs.push_back(packed.data_ptr<int64_t>() + offset);
    offset += vec.size();
  }
  return std::make_pair(packed, ptrs);
}

void fsdpAllGatherCopyOut(
    std::vector<at::Tensor> params,
    at::Tensor allGatherRes,
    int64_t worldSize) {
  const auto device = allGatherRes.device();
  const auto totalSize = allGatherRes.numel() * allGatherRes.element_size();

  TORCH_CHECK(allGatherRes.is_cuda());
  TORCH_CHECK(allGatherRes.is_non_overlapping_and_dense());

  std::vector<int64_t> paramPtrs;
  std::vector<int64_t> shardDims; // In bytes
  std::vector<int64_t> dimCumSums{0}; // In bytes
  for (size_t i = 0; i < params.size(); ++i) {
    const auto& param = params[i];
    TORCH_CHECK(param.is_non_overlapping_and_dense());
    TORCH_CHECK(param.device() == device);
    TORCH_CHECK(param.numel() > 0);
    // All params are expected to be aligned at worldSize.
    TORCH_CHECK(param.numel() % worldSize == 0);
    const auto shardDim = param.numel() * param.element_size() / worldSize;
    paramPtrs.push_back(reinterpret_cast<int64_t>(param.data_ptr()));
    shardDims.push_back(shardDim);
    dimCumSums.push_back(dimCumSums[i] + shardDim);
  }

  TORCH_CHECK(
      dimCumSums.back() * worldSize == totalSize,
      "The total byte size must be identical between params and allGatherRes");

  // To balance the throughput larger shards and waste on smaller shards, we
  // use the geometric mean of the shard dims to determine the block size.
  int64_t meanShardDim = geometricMean(shardDims);
  int64_t blockSize = divUp(meanShardDim, BYTES_PER_THREAD);
  blockSize = divUp(blockSize, WARP_SIZE) * WARP_SIZE;
  blockSize = std::min(std::max(blockSize, MIN_NUM_THREADS), MAX_NUM_THREADS);

  // TODO: this is only for A100
  constexpr int64_t maxActiveBlocks = 32 * 108;
  constexpr double smOverSubFactor = 1.75;

  // Roughly estimate the amount of blocks needed for each rank, and calculate
  // an iter factor to regularize SM over-subscription.
  int64_t iterFactor = 1;
  while (divUp(totalSize, blockSize * BYTES_PER_THREAD * iterFactor) >
         (maxActiveBlocks * smOverSubFactor)) {
    iterFactor += 1;
  }

  std::vector<int64_t> blockOffsetToParamIdx;
  std::vector<int64_t> blockCumSums{0};
  for (int64_t paramIdx = 0; paramIdx < static_cast<int64_t>(params.size());
       ++paramIdx) {
    int64_t numBlocks =
        divUp(shardDims[paramIdx], blockSize * BYTES_PER_THREAD * iterFactor);
    blockOffsetToParamIdx.insert(
        blockOffsetToParamIdx.end(), numBlocks, paramIdx);
    blockCumSums.push_back(blockCumSums.back() + numBlocks);
  }
  const auto numBlocks = blockCumSums.back();

  auto packed = pack(
      {paramPtrs, blockOffsetToParamIdx, blockCumSums, dimCumSums}, device);

  int64_t ranksPerBlock = 1;
  while (numBlocks * (worldSize / ranksPerBlock) >
             maxActiveBlocks * smOverSubFactor &&
         ranksPerBlock < worldSize) {
    ++ranksPerBlock;
  }

  dim3 blocks(numBlocks * (worldSize / ranksPerBlock), 1, 1);
  dim3 threads(blockSize, 1, 1);

  LOG(INFO) << "meanShardDim: " << meanShardDim
            << ", iterFactor: " << iterFactor
            << ", ranksPerBlock: " << ranksPerBlock << ", blocks: " << blocks.x
            << ", threads: " << threads.x;

  fsdpAllGatherCopyOutKernel<<<
      blocks,
      threads,
      0,
      at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<void**>(packed.second[0]),
      allGatherRes.data_ptr(),
      /*blockOffsetToParamIdx=*/packed.second[1],
      /*blockCumSums=*/packed.second[2],
      /*shardDimCumSums=*/packed.second[3],
      dimCumSums.back(),
      blockCumSums.back(),
      worldSize / ranksPerBlock,
      worldSize);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

static __global__ void padCatDim0Kernel(
  void **tensors,
  int64_t factor,
  void* out,
  int64_t* blockOffsetToTensorIdx,
  int64_t* cumSumBlocksPerShard,
  int64_t* cumSumNumBytesPerShard,
  int64_t numBytesPerRank,
  int64_t numBlocksPerRank,
  int64_t* tensorBytes,
  int64_t rankStride
) {
  const int64_t blockOffset = blockIdx.x % numBlocksPerRank;
  const int64_t tensorIdx = blockOffsetToTensorIdx[blockOffset];
  for (int64_t rank = blockIdx.x / numBlocksPerRank; rank < factor; rank += rankStride) {
    const int64_t shardBlockCount = cumSumBlocksPerShard[tensorIdx + 1] - cumSumBlocksPerShard[tensorIdx];
    const int64_t groupSize = shardBlockCount * blockDim.x;
    const int64_t localTid = (blockOffset - cumSumBlocksPerShard[tensorIdx]) * blockDim.x + threadIdx.x;
    const int64_t shardBegin = cumSumNumBytesPerShard[tensorIdx];
    const int64_t shardEnd = cumSumNumBytesPerShard[tensorIdx+1];
    const int64_t theoryShardNumBytes = shardEnd - shardBegin;
    const int64_t actualNumBytes = minInt64(theoryShardNumBytes, maxInt64(tensorBytes[tensorIdx] - rank * theoryShardNumBytes, 0));
    const int64_t dstOff = rank * numBytesPerRank + shardBegin;
    const int64_t srcOff = rank * theoryShardNumBytes;
    char* dstPtr = reinterpret_cast<char*>(out) + dstOff;
    const char* srcPtr = reinterpret_cast<char*>(tensors[tensorIdx]) + srcOff;
    const int64_t alignOff =
      divUp(dstOff, BYTES_PER_THREAD) * BYTES_PER_THREAD - dstOff;
    const int64_t begin = alignOff + localTid * BYTES_PER_THREAD;
    const int64_t end = alignOff + (actualNumBytes - alignOff) / BYTES_PER_THREAD * BYTES_PER_THREAD;
    const int64_t stride = groupSize * BYTES_PER_THREAD;
    const uint4 zero = initialize();
    for (size_t i = begin; i < end; i += stride) {
      uint4 val = zero;
      if(isAligned(srcPtr + i, BYTES_PER_THREAD)) {
        streamLoad128(val, srcPtr + i);
      } else {
        for (size_t j = 0; j < BYTES_PER_THREAD; ++j) {
          reinterpret_cast<char*>(&val)[j] = srcPtr[i + j];
        }
      }
      streamStore128(&dstPtr[i], val);
    }
    if(localTid < alignOff && localTid < theoryShardNumBytes) {
      char val = (char) 0;
      if (localTid < actualNumBytes) {
        val = srcPtr[localTid];
      }
      dstPtr[localTid] = val;
    }
    if(end + localTid < theoryShardNumBytes) {
      char val = (char) 0;
      if (end + localTid < actualNumBytes) {
        val = srcPtr[end + localTid];
      }
      dstPtr[end + localTid] = val;
    }
  }
}

// Pad and cat along 0-th dimension. We do not assume that param.numel() % factor == 0.
// TODO: Add more doc.
void padCatDim0(
  std::vector<at::Tensor> tensors,
  int64_t factor,
  at::Tensor out
) {
  const auto device = out.device();
  const auto totalSize = out.numel() * out.element_size();
  TORCH_CHECK(out.is_cuda());
  TORCH_CHECK(out.is_non_overlapping_and_dense());
  std::vector<int64_t> tensorPtrs;
  std::vector<int64_t> tensorBytes;
  std::vector<int64_t> tensorIdxToNumBytesPerShard;
  std::vector<int64_t> cumSumNumBytesPerShard{0};
  // TODO: We may only do boundary check once.
  // There are three types of blocks: within boundary, outside boundary, or on the boundary. We do not need to check boundary many times.
  for (size_t i = 0; i < tensors.size(); ++i) {
    const auto& tensor = tensors[i];
    TORCH_CHECK(tensor.is_non_overlapping_and_dense());
    TORCH_CHECK(tensor.device() == device);
    TORCH_CHECK(tensor.numel() > 0);
    const auto sizes = tensor.sizes();
    const int64_t sizeRemainingDims = tensor.numel() / sizes[0];
    const int64_t padSizeAlongDim = divUp(sizes[0], factor) * factor;
    const int64_t numBytesPerShard = padSizeAlongDim * sizeRemainingDims * tensor.element_size() / factor;
    tensorPtrs.push_back(reinterpret_cast<int64_t>(tensor.data_ptr()));
    tensorBytes.push_back(tensor.numel() * tensor.element_size());
    tensorIdxToNumBytesPerShard.push_back(numBytesPerShard);
    cumSumNumBytesPerShard.push_back(cumSumNumBytesPerShard[i] + numBytesPerShard);
  }
  constexpr int64_t maxActiveBlocks = 32 * 132;
  constexpr int64_t smOverSubFactor = 1.75;
  std::vector<int64_t> blockOffsetToTensorIdx;
  std::vector<int64_t> cumSumBlocksPerShard{0};
  for (int64_t tensorIdx = 0; tensorIdx < static_cast<int64_t>(tensors.size()); ++tensorIdx) {
    int64_t numBlocksPerShard = divUp(tensorIdxToNumBytesPerShard[tensorIdx], BLOCK_SIZE * BYTES_PER_THREAD);
    blockOffsetToTensorIdx.insert(blockOffsetToTensorIdx.end(), numBlocksPerShard, tensorIdx);
    cumSumBlocksPerShard.push_back(cumSumBlocksPerShard.back() + numBlocksPerShard);
  }
  const auto numBlocksPerRank = cumSumBlocksPerShard.back();
  auto packed = pack(
    {tensorPtrs, blockOffsetToTensorIdx, cumSumBlocksPerShard, cumSumNumBytesPerShard, tensorBytes}, device
  );
  int64_t ranksPerBlock = 1;
  while (numBlocksPerRank * (factor / ranksPerBlock) >
          maxActiveBlocks * smOverSubFactor &&
        ranksPerBlock < factor) {
    ++ranksPerBlock;
  }
  dim3 blocks(numBlocksPerRank * (factor / ranksPerBlock), 1, 1);
  dim3 threads(BLOCK_SIZE, 1, 1);
  padCatDim0Kernel<<<
    blocks,
    threads,
    0,
    at::cuda::getCurrentCUDAStream()>>>(
      reinterpret_cast<void**>(packed.second[0]),
      factor,
      out.data_ptr(),
      /*blockOffsetToTensorIdx=*/packed.second[1],
      /*cumSumBlocksPerShard=*/packed.second[2],
      /*cumSumNumBytesPerShard=*/packed.second[3],
      cumSumNumBytesPerShard.back(),
      numBlocksPerRank,
      /*tensorBytes=*/packed.second[4],
      factor / ranksPerBlock
  );
}

// TODO
void padCat(
  std::vector<at::Tensor> tensors,
  int64_t dim,
  int64_t factor,
  at::Tensor out
) {
  TORCH_CHECK(dim == 0);
  padCatDim0(tensors, factor, out);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
